#include "hip/hip_runtime.h"
/*
------------------DR VASILIOS KELEFOURAS-----------------------------------------------------
------------------COMP3001 ------------------------------------------------------------------
------------------PARALLEL PROGAMMING MODULE-------------------------------------------------
------------------UNIVERSITY OF PLYMOUTH, SCHOOL OF ENGINEERING, COMPUTING AND MATHEMATICS---
*/

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <omp.h>


#define N 256 //input size


__declspec(align(32)) float test[N][N][N], sum[N][N][N], A[N][N][N], C[N][N]; 

__device__ float device_sum[N][N][N], device_A[N][N][N], device_C[N][N]; //allocate the device arrays statically (global GPU memory)

void init();
void default();
int Compare();
inline unsigned short int equal(float const a, float const b);


#define EPSILON 0.00001

#define MAX_NUMBER_OF_BLOCKS_PER_DIM 65535 //max number of blocks that our GPU can handle (for one dimension only)


__global__ void diotgen_ver1() {

//write your code here
/*
	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
			for (int p = 0; p < N; p++)
				for (int s = 0; s < N; s++)
					sum[r][q][p] = sum[r][q][p] + A[r][q][s] * C[s][p];
*/


	dim3 dimGrid(128, 1, 1);
	dim3 dimBlock(N / 128, 1, 1);
	dim3 dimThreads(N / 128, 1, 1);


	int g_id = threadIdx.x + blockIdx.x * blockDim.x;

	if (g_id < N) {
		printf("\n global %d | thread %d | block %d\n", g_id, threadIdx.x, blockIdx.x);
	}



}

int main()
{
	hipError_t cudaStatus;

	//------create the cuda timers------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;

	int devId = 0;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devId);
	printf("\n Device: %s \n", prop.name);

	init(); //initialize host arrays


	/* Copy the A array from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(device_A), A, N * N *N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}

	/* Copy the C array from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(device_C), C, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}

	hipEventRecord(start, 0); //get timer value


		dim3 dimBlock(1,1 ,1 );
		dim3 dimGrid(1 ,1 ,1 );
		diotgen_ver1 << <dimGrid, dimBlock >> > ( );
		


	hipEventRecord(stop, 0);  //get timer value
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("\nElapsed time in msecs = %f", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	/* Copy back the result from the DEVICE memory to the HOST memory */
	cudaStatus = hipMemcpyFromSymbol(sum, HIP_SYMBOL(device_sum), N * N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}

	//do not forget to print the flops value achieved

	/*  Handling function of the CUDA runtime application programming interface.
	*   Returns the last error from a runtime call.
	*/
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}



	if (Compare() != 0)
		printf("\n---------WRONG OUTPUT---------------\n");
	else
		printf("\n---------OUTPUT IS OK---------------\n");


	/* Destroy all allocations and reset all state on the current device in the current process */
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("\ncuda Reset failed!");
		return -1;
	}

	return 0;
}


void init() {

	float e = 0.12, p = 0.72;
	unsigned int i, j, k;

	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			C[i][j] = (j % 9) + p;
		}
	}

	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			for (k = 0; k < N; k++) {
				sum[i][j][k] = 0.0;
				test[i][j][k] = 0.0;
				A[i][j][k] = (((i + j) % 99) + e);
			}
		}
	}


}

//this is the routine that you will parallelize 
void default() {

	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
			for (int p = 0; p < N; p++)
				for (int s = 0; s < N; s++)
					test[r][q][p] = test[r][q][p] + A[r][q][s] * C[s][p];


}


unsigned short int equal(float const a, float const b) {
	float temp = a - b;
	//printf("\n %f  %f", a, b);
	if (fabs(temp/b) < EPSILON)
		return 0; //success
	else
		return 1;
}


int Compare() {


	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
			for (int p = 0; p < N; p++)
				for (int s = 0; s < N; s++)
					test[r][q][p] = test[r][q][p] + A[r][q][s] * C[s][p];


	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
				for (int p = 0; p < N; p++)
					if (equal(sum[r][q][p], test[r][q][p]) == 1) {
				      printf("\n wrong at (%d,%d,%d)", r, q,p);
					  return -1;
					}
	return 0;
}





