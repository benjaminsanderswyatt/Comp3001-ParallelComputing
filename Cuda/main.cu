#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void helloWorld() {
	int global_index = blockIdx.x * blockDim.x + threadIdx.x;
	printf("\n Hello from thread %d and block %d meaning global %d", threadIdx.x, blockIdx.x, global_index);
	

}



int main() {

	dim3 blocks(1, 1, 1); // grid size and dimensions, it consists of blocks
	dim3 threads(6, 1, 1); // block size and dimensions, it consists of threads


	helloWorld <<<blocks, threads >>> (); // MaxBlocks: , MaxThreads: 1024

	// Error message
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}



	hipDeviceReset();

	return 0;
}