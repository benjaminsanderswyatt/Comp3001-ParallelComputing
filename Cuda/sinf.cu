#include "hip/hip_runtime.h"
/*
------------------DR VASILIOS KELEFOURAS-----------------------------------------------------
------------------COMP3001 ------------------------------------------------------------------
------------------PARALLEL PROGAMMING MODULE-------------------------------------------------
------------------UNIVERSITY OF PLYMOUTH, SCHOOL OF ENGINEERING, COMPUTING AND MATHEMATICS---
*/


#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define N 100 //input size

void initialization(float*in, float*out);
void sin_serial(const float* in, float* out);
void print_arrays(const float* in, const float* out);
int compare(const float* in, float* out);
__global__ void sin_parallel(const float* in, float* out); //CUDA kernel - this function will run on the GPU


int main(){

   //the arrays are allocated dynamically in this case. They can also be allocated statically (we will see that next week)
    float* input, * output;//these are the host arrays
    float* d_input, * d_output;//these are the device arrays

    hipError_t cudaStatus;

    input = (float*)malloc(N * sizeof(float)); //dynamically allocate CPU memory using malloc
    if (input == NULL) { //if memory asked cannot be allocated, e.g., too large
        printf("\nmemory did not allocated\n");
        return -1;
    }


    output = (float*)malloc(N * sizeof(float)); //dynamically allocate CPU memory using malloc
    if (output == NULL) {
        printf("\nmemory did not allocated\n");
        free(input);
        return -1;
    }

    //initialize the host arrays
    initialization(input, output);

    cudaStatus=hipMalloc(&d_input, N * sizeof(float));//dynamically allocate memory in GPU by using cudamalloc
    if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available 
        printf("\ncudaMalloc failed!");
        free(input); free(output); //free the already allocated arrays
        return -1; //end the process
    }

    cudaStatus=hipMalloc(&d_output, N * sizeof(float));//dynamically allocate memory in GPU by using cudamalloc
    if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available 
        printf("\ncudaMalloc failed!");
        free(input); free(output); hipFree(d_input); //free the already allocated arrays
        return -1;//end the process
    }

  

    cudaStatus=hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);//copy data from the host array to the device array
    if (cudaStatus != hipSuccess) {//if the array cannot be copied
        printf("\ncudaMemcpy failed!");
        free(input); free(output); hipFree(d_input); hipFree(d_output); //free the already allocated arrays
        return -1;//end the process
    }


    // Use a number of threads that is either 128, 256, 512, 1024 (we will explain that later)
    dim3 dimGrid(128, 1, 1);//1d grid consists of 10 blocks. x dim is the first (x, y, z )
    // Use a number of blocks given by N / num.threads
    dim3 dimBlock(2, 1, 1); //1d blocks consisting of 10 threads. x dim is the first (x, y, z )


    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);
    int numThreads = deviceProperties.maxThreadsPerBlock;
    printf("\n % d", numThreads);

    

    
    sin_parallel << <dimBlock, dimGrid >> > (d_input, d_output); //launch the kernel

    hipError_t error = hipGetLastError(); //get the status of the last cuda kernel that was called (in this case this is the sin_parallel)
    if (error != hipSuccess) //if the sin_parallel() function did not run sucessfully
        printf("\nError %s\n",hipGetErrorString(error)); //use this function to show the description of the error

    cudaStatus=hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);//copy data from the device array to host array
    if (cudaStatus != hipSuccess) {//if the array cannot be copied
        printf("\ncudaMemcpy failed!");
        free(input); free(output); hipFree(d_input); hipFree(d_output);
        return -1;
    }

    
    //compare the output of sin_parallel to verify that the CUDA kernel works fine
    compare(input, output);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    /*hipDeviceReset() Explicitly destroys and cleans up all resources associated with the current device in the 
    current process. Any subsequent API call to this device will reinitialize the device.
    Note that this function will reset the device immediately.It is the caller's responsibility 
    to ensure that the device is not being accessed by any other host threads from the process 
    when this function is called. */

 
    //print_arrays(input, output);

    //sin_serial(input, output);
    //print_arrays(input, output);

    free(input);//free the memory allocated dynamically in the CPU
    free(output); //free the memory allocated dynamically in the CPU
    hipFree(d_input);//free the memory allocated dynamically in the GPU
    hipFree(d_output);//free the memory allocated dynamically in the GPU

    return 0; //exit sucessfully
}


void initialization(float* in, float* out) {

    int i;
    for (i = 0; i < N; i++) {
        in[i] = (float)(rand() / 7.1f);
        out[i] = 0.0f;
    }

}

void print_arrays(const float* in, const float* out) {

    int i;
    for (i = 0; i < N; i++)
        printf("\ninput, output are %f , %f\n",in[i],out[i]);

    printf("\n\n");

}


void sin_serial(const float* in, float* out) {

    int i;
    for (i = 0; i < N; i++)
        out[i] = sinf(in[i]);

}


int compare(const float* in, float* out) {

    int i;
    for (i = 0; i < N; i++) {
        if ( fabs((out[i] - sinf(in[i])) / out[i]) > 0.00001 ) {
            printf("\n\wrong results %f - %f\n", out[i], sinf(in[i]));
            return -1;
        }
    }
    printf("\nResults are correct\n");
    return 0;
}


__global__ void sin_parallel(const float* in, float* out) {
    int g_id = threadIdx.x + blockIdx.x * blockDim.x;

    if (g_id < N) {
        out[g_id] = sinf(in[g_id]);
        printf("\n global %d | thread %d | block %d | %f %f\n",g_id, threadIdx.x, blockIdx.x, in[g_id], out[g_id]);
    }
    
}

