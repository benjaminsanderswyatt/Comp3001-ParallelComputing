#include <hip/hip_runtime.h> 
 

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define N 64  //arrays input size
#define TIMES 1 //times to run
#define ARITHMETICAL_OPS N*N*N*2

#define EPSILON 0.00001


__declspec(align(64)) float test[N*N], A[N*N], B[N*N], C[N*N];




unsigned short int compare(const float* C, const float* A, const float* B);
unsigned short int equal(float const x, float const y);

void init(); // Init

void cuda_error();


// --------------------------------------- implementation #3 ---------------------------------------
__global__ void implementation_3(float* C, float* A, float* B) {

	printf("\nHello from thread %d of block %d", threadIdx.x, blockIdx.x);

	float tmp = 0.0;



	int i = blockIdx.x * blockDim.x + threadIdx.x; //i loop has been parallelized 

	int j = blockIdx.y * blockDim.y + threadIdx.y; //j loop has been parallelized 



	for (int k = 0; k < N; k++) {

		tmp += A[N * i + k] * B[N * k + j];

	}



	C[N * i + j] = tmp;
}




int main() {
	printf("\n------------- implementation #3 -------------\n");

	hipError_t cudaStatus;

	//------create the cuda timers------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;

	int devId = 0;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devId);
	printf("\n Device: %s \n", prop.name);

	init(); //initialize host arrays

	float* C_d, * A_d, * B_d;

	// create GPU arrays
	cudaStatus = hipMalloc((void**)&C_d, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\nCudaMalloc failed");
		hipFree(C_d);
		return -1;
	}

	cudaStatus = hipMalloc((void**)&A_d, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\nCudaMalloc failed");
		hipFree(C_d); hipFree(A_d);
		return -1;
	}

	cudaStatus = hipMalloc((void**)&B_d, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\nCudaMalloc failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;
	}


	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid(N / 16, N / 16, 1);



	hipEventRecord(start, 0); //get timer value


	// copy arrays from host to device
	cudaStatus = hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice); //copy array from host to GPU
	if (cudaStatus != hipSuccess) {
		printf("\ncuda copy failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;
	}

	cudaStatus = hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice); //copy array from host to GPU
	if (cudaStatus != hipSuccess) {
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		printf("\ncuda copy failed");
		return -1;
	}


	for (int i = 0; i < TIMES; i++) {
		implementation_3 <<<dimGrid, dimBlock>>> (C,A,B);

	}


	cudaStatus = hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost); //copy array from GPU back to CPU
	if (cudaStatus != hipSuccess) {
		printf("\ncuda copy failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;
	}


	hipEventRecord(stop, 0);  //get timer value
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("\nElapsed time in msecs = %f", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	double flops = (double)((double)2 * N * N * N) / (elapsed_time / TIMES);
	printf("\nGflops achieved %f ", flops / 1000000);

	cuda_error();

	if (compare(C,A,B) == 0)
		printf("\nResult is ok\n");
	else
		printf("\nResult is FALSE\n");

	hipDeviceReset();

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("\ncuda Reset failed!");
		return -1;
	}

	return 0;
}

// CUDA error
void cuda_error() {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("\nError: %s\n", hipGetErrorString(error));
	}
}



// Init
void init() {

	int i;

	for (i = 0; i < (N * N); i++) {
		A[i] = (float)i;
		B[i] = (float)i + 1;
		C[i] = 0.0f;
	}

}



unsigned short int compare(const float* C, const float* A, const float* B) {

	int i, j, k;

	for (i = 0; i < (N * N); i++) {
		test[i] = 0.0f;
	}


	for (i = 0; i < N; i++)
		for (j = 0; j < N; j++)
			for (k = 0; k < N; k++)
				test[N * i + j] += A[N * i + k] * B[N * k + j];

	for (j = 0; j < (N * N); j++)
		if (equal(C[j], test[j]) == 1) {
			printf("\n j=%d %f %f\n", j, test[j], C[j]);
			return -1;
		}

	return 0;
}


unsigned short int equal(float const x, float const y) {
	float temp = x - y;
	//printf("\n %f  %f", a, b);
	if (fabs(temp / y) < EPSILON)
		return 0; //success
	else
		return 1;
}