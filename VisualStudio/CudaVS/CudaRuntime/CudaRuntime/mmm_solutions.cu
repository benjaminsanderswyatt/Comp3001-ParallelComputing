#include "hip/hip_runtime.h"
/*
------------------DR VASILIOS KELEFOURAS-----------------------------------------------------
------------------COMP3001 ------------------------------------------------------------------
------------------PARALLEL PROGAMMING MODULE-------------------------------------------------
------------------UNIVERSITY OF PLYMOUTH, SCHOOL OF ENGINEERING, COMPUTING AND MATHEMATICS---
*/

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <omp.h>

#define TIMES_TO_RUN 1 //how many times the function will run

#define N 1024 //input size - USE POWER OF 2 ONLY 
#define CHECK_OUTPUT   //if do not want to validate the results comment this 

#define TILE 32 //use either 32 or 16 only
#define TILE_x2 TILE*2
#define TILE_x4 TILE*4

__declspec(align(64)) float C[N * N], test[N * N], A[N * N], B[N * N]; //square matrixes are considered only, stored as 1d arrays

void MMM_init();
void MMM_default();
int Compare_MMM();
inline unsigned short int equal(float const a, float const b);


#define EPSILON 0.00001

#define MAX_NUMBER_OF_BLOCKS_PER_DIM 65535 //max number of blocks that our GPU can handle (for one dimension only)




//This implementation uses a 2d grid and 2d blocks of threads.
//each thread computes a value in C[], thus N*N threads
__global__ void mmm_ver1(float* C, float* A, float* B) {



}


int main()
{
	hipError_t cudaStatus;

	//------create the cuda timers------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;

	int devId = 0;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devId);
	printf("\n Device: %s \n", prop.name);

	MMM_init(); //initialize host arrays

	float* C_d, * A_d, * B_d; //pointers to device arrays

//---------------------------create GPU arrays------------------------------------------
	cudaStatus = hipMalloc((void**)&C_d, N * N * sizeof(float));//allocate memory dynamically 
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available
		printf("\nCudaMalloc failed");
		hipFree(C_d);
		return -1;//returns unsuccessfully
	}

	cudaStatus = hipMalloc((void**)&A_d, N * N * sizeof(float));//allocate memory dynamically 
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available
		printf("\nCudaMalloc failed");
		hipFree(C_d); hipFree(A_d);
		return -1;//returns unsuccessfully
	}

	cudaStatus = hipMalloc((void**)&B_d, N * N * sizeof(float));//allocate memory dynamically 
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available
		printf("\nCudaMalloc failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;//returns unsuccessfully
	}



	//--------------------copy arrays from host to device------------------------
	cudaStatus = hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice); //copy array from host to GPU
	if (cudaStatus != hipSuccess) {//if cuda copy fails
		printf("\ncuda copy failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;//returns unsuccessfully
	}

	cudaStatus = hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice); //copy array from host to GPU
	if (cudaStatus != hipSuccess) {//if cuda copy fails
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		printf("\ncuda copy failed");
		return -1;//returns unsuccessfully
	}


	hipEventRecord(start, 0); //get timer value

	for (int it = 0; it < TIMES_TO_RUN; it++) { //in CUDA you do not have to run the program many times to get an accurate ex.time. However, the first time you run a kernel it normally takes more. So you need to include one - two extra runs (this is also known as warm up time). The CUDA timers have an accuracy of 1 msec.

		//dim3 dimBlock(TILE, TILE, 1);
		//dim3 dimGrid((N + TILE - 1) / TILE, (N + TILE - 1) / TILE, 1);
		//mmm_ver1 << <dimGrid, dimBlock >> > (C_d, A_d, B_d);
		//mmm_tiled << <dimGrid, dimBlock >> > (C_d, A_d, B_d);
		//mmm_sw_pipeline <<< dimGrid, dimBlock >>> (C_d, A_d, B_d);

		//for mmm_tiled_regblocking_factor2() use the following parameters
		dim3 dimBlock(TILE, TILE, 1);
		dim3 dimGrid((N + TILE_x2 - 1) / (TILE_x2), (N + TILE_x2 - 1) / (TILE_x2), 1);
		mmm_ver1 << <dimGrid, dimBlock >> > (C_d, A_d, B_d);

	}


	hipEventRecord(stop, 0);  //get timer value
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("\nElapsed time in msecs = %f", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	double flops = (double)((double)2 * N * N * N) / (elapsed_time / TIMES_TO_RUN);
	printf("\nGflops achieved %f ", flops / 1000000);

	/*  Handling function of the CUDA runtime application programming interface.
	*   Returns the last error from a runtime call.
	*/
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}


	cudaStatus = hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost); //copy array from GPU back to CPU
	if (cudaStatus != hipSuccess) {//if cuda copy fails
		printf("\ncuda copy failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;//returns unsuccessfully
	}

	//MMM_default();

#ifdef CHECK_OUTPUT 
	if (Compare_MMM() != 0)
		printf("\n---------WRONG OUTPUT---------------\n");
	else
		printf("\n---------OUTPUT IS OK---------------\n");
#endif

	/* Destroy all allocations and reset all state on the current device in the current process */
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("\ncuda Reset failed!");
		return -1;
	}

	return 0;
}


void MMM_init() {

	float e = 0.1234, p = 0.7264, r = 0.11;

	//MMM
	for (unsigned int i = 0; i < N; i++) { //printf("\n");
		for (unsigned int j = 0; j < N; j++) {
			C[N * i + j] = 0.0;
			test[N * i + j] = 0.0;
			A[N * i + j] = (j % 9) + p; //printf(" %3.1f",A[i][j]);
			B[N * i + j] = (j % 7) - p; //printf(" %3.1f",B[i][j]);
		}
	}


}


void MMM_default() {

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			for (int k = 0; k < N; k++)
				C[N * i + j] += A[N * i + k] * B[N * k + j];


}


unsigned short int equal(float const a, float const b) {
	float temp = a - b;
	//printf("\n %f  %f", a, b);
	if (fabs(temp/b) < EPSILON)
		return 0; //success
	else
		return 1;
}


int Compare_MMM() {

	float tmp;
	int i, j, k;

	//optimize the following, otherwise it takes too long...however, to allow VS to use the \pragmas you must go 
		//in project  properties and enable that (look at the lab session document for more info)
#pragma omp parallel 
	{
#pragma omp for private(i, j, k, tmp)
		for (i = 0; i < N; i++) {
			for (j = 0; j < N; j++) {
				tmp = 0.0;
#pragma omp simd reduction(+:tmp) aligned(C,A,B:64)
				for (k = 0; k < N; k++) {
					tmp += A[N * i + k] * B[N * k + j];
				}
				test[N * i + j] = tmp;
			}
		}
	}

	for (i = 0; i < N; i++)
		for (j = 0; j < N; j++)
			if (equal(C[N * i + j], test[N * i + j]) == 1) {
				printf("\n wrong at (%d,%d) - %f %f", i, j, C[N * i + j], test[N * i + j]);
				return -1;
			}
	return 0;
}





